/*
 * Copyright (C) 2020 Jordan Hendl
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include "Cuda.h"
#define VULKAN_HPP_NO_EXCEPTIONS
#include <algorithm>

#ifdef KGL_CUDA_FOUND // Won't get compiler errors when building this without vulkan.


 namespace kgl
 {
   namespace cg
   {
     void Cuda::initialize()
     {

     }

     unsigned Cuda::convertError( unsigned error )
     {
       return 0 ;
     }

     void Cuda::copyTo( const void* src, Cuda::Memory& dst, Cuda::Device& gpu, unsigned amt )
     {
       hipSetDevice( gpu ) ;
       hipMemcpy( dst, src, amt, hipMemcpyHostToDevice ) ;
     }
     
     void Cuda::copyTo( const Cuda::Memory& src, Cuda::Data dst, Cuda::Device& gpu, unsigned amt )
     {
       hipSetDevice( gpu ) ;
       hipMemcpy( dst, src, amt, hipMemcpyDeviceToHost ) ;
     }
     
     void Cuda::free( Cuda::Memory& mem, Cuda::Device& gpu )
     {
       hipSetDevice( gpu ) ;
       hipFree( mem ) ;
     }
     
     Cuda::Memory Cuda::createMemory( unsigned size, const Cuda::Device& gpu, Cuda::MemoryFlags flags )
     {
       return Cuda::createMemory( size, gpu ) ;
     }
     
     Cuda::Memory Cuda::createMemory( unsigned size, const Cuda::Device& gpu )
     {
       unsigned char* data ;

       hipSetDevice( gpu         ) ;
       hipMalloc   ( &data, size ) ;
       
       return data ;
     }
   }
 }
#endif
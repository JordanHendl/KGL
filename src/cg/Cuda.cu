/*
 * Copyright (C) 2020 Jordan Hendl
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include "Cuda.h"
#define VULKAN_HPP_NO_EXCEPTIONS
#include <algorithm>
#include <iostream>

namespace kgl
{
  namespace cg
  {
    /** Function to handle the input cuda error.
     * @param error The error to handle by the library.
     */
    static void handleError( hipError_t error ) ;
    
    
    void handleError( hipError_t error )
    {
      switch( error )
      {
        case hipSuccess                           :
        case hipErrorInvalidValue                 :
        case hipErrorOutOfMemory             :
        case hipErrorNotInitialized          :
        case hipErrorDeinitialized              :
        case hipErrorProfilerDisabled             :
        case hipErrorProfilerNotInitialized       :
        case hipErrorProfilerAlreadyStarted       :
        case hipErrorProfilerAlreadyStopped       :
        case hipErrorInvalidConfiguration         :
        case hipErrorInvalidPitchValue            :
        case hipErrorInvalidSymbol                :
        case cudaErrorInvalidHostPointer           :
        case hipErrorInvalidDevicePointer         :
        case hipErrorInvalidTexture               :
        case cudaErrorInvalidTextureBinding        :
        case hipErrorInvalidChannelDescriptor     :
        case hipErrorInvalidMemcpyDirection       :
        case cudaErrorAddressOfConstant            :
        case cudaErrorTextureFetchFailed           :
        case cudaErrorTextureNotBound              :
        case cudaErrorSynchronizationError         :
        case cudaErrorInvalidFilterSetting         :
        case cudaErrorInvalidNormSetting           :
        case cudaErrorMixedDeviceExecution         :
        case cudaErrorNotYetImplemented            :
        case cudaErrorMemoryValueTooLarge          :
        case hipErrorInsufficientDriver           :
        case cudaErrorInvalidSurface               :
        case cudaErrorDuplicateVariableName        :
        case cudaErrorDuplicateTextureName         :
        case cudaErrorDuplicateSurfaceName         :
        case cudaErrorDevicesUnavailable           :
        case cudaErrorIncompatibleDriverContext    :
        case hipErrorMissingConfiguration         :
        case hipErrorPriorLaunchFailure           :
        case cudaErrorLaunchMaxDepthExceeded       :
        case cudaErrorLaunchFileScopedTex          :
        case cudaErrorLaunchFileScopedSurf         :
        case cudaErrorSyncDepthExceeded            :
        case cudaErrorLaunchPendingCountExceeded   :
        case hipErrorInvalidDeviceFunction        :
        case hipErrorNoDevice                     :
        case hipErrorInvalidDevice                :
        case cudaErrorStartupFailure               :
        case hipErrorInvalidImage           :
        case hipErrorInvalidContext          :
        case hipErrorMapFailed        :
        case hipErrorUnmapFailed      :
        case hipErrorArrayIsMapped                :
        case hipErrorAlreadyMapped                :
        case hipErrorNoBinaryForGpu       :
        case hipErrorAlreadyAcquired              :
        case hipErrorNotMapped                    :
        case hipErrorNotMappedAsArray             :
        case hipErrorNotMappedAsPointer           :
        case hipErrorECCNotCorrectable             :
        case hipErrorUnsupportedLimit             :
        case hipErrorContextAlreadyInUse           :
        case hipErrorPeerAccessUnsupported        :
        case hipErrorInvalidKernelFile                   :
        case hipErrorInvalidGraphicsContext       :
        case cudaErrorNvlinkUncorrectable          :
        case cudaErrorJitCompilerNotFound          :
        case hipErrorInvalidSource                :
        case hipErrorFileNotFound                 :
        case hipErrorSharedObjectSymbolNotFound   :
        case hipErrorSharedObjectInitFailed       :
        case hipErrorOperatingSystem              :
        case hipErrorInvalidHandle        :
        case hipErrorIllegalState                 :
        case hipErrorNotFound               :
        case hipErrorNotReady                     :
        case hipErrorIllegalAddress               :
        case hipErrorLaunchOutOfResources         :
        case hipErrorLaunchTimeOut                :
        case cudaErrorLaunchIncompatibleTexturing  :
        case hipErrorPeerAccessAlreadyEnabled     :
        case hipErrorPeerAccessNotEnabled         :
        case hipErrorSetOnActiveProcess           :
        case hipErrorContextIsDestroyed           :
        case hipErrorAssert                       :
        case cudaErrorTooManyPeers                 :
        case hipErrorHostMemoryAlreadyRegistered  :
        case hipErrorHostMemoryNotRegistered      :
        case cudaErrorHardwareStackError           :
        case cudaErrorIllegalInstruction           :
        case cudaErrorMisalignedAddress            :
        case cudaErrorInvalidAddressSpace          :
        case cudaErrorInvalidPc                    :
        case hipErrorLaunchFailure                :
        case hipErrorCooperativeLaunchTooLarge    :
        case cudaErrorNotPermitted                 :
        case hipErrorNotSupported                 :
        case cudaErrorSystemNotReady               :
        case cudaErrorSystemDriverMismatch         :
        case cudaErrorCompatNotSupportedOnDevice   :
        case hipErrorStreamCaptureUnsupported     :
        case hipErrorStreamCaptureInvalidated     :
        case hipErrorStreamCaptureMerge           :
        case hipErrorStreamCaptureUnmatched       :
        case hipErrorStreamCaptureUnjoined        :
        case hipErrorStreamCaptureIsolation       :
        case hipErrorStreamCaptureImplicit        :
        case hipErrorCapturedEvent                :
        case hipErrorStreamCaptureWrongThread     :
        case cudaErrorTimeout                      :
        case hipErrorGraphExecUpdateFailure       :
        case hipErrorUnknown                      :
        case cudaErrorApiFailureBase               :
          
          std::cout << "Error handling for Cuda Library not implemented yet." << std::endl ;
          break ;
      }
    }

    void Cuda::initialize()
    {

    }

    unsigned Cuda::convertError( unsigned error )
    {
      return 0 ;
    }

    void Cuda::copyToDevice( const void* src, Cuda::Memory& dst, Cuda::Device& gpu, unsigned amt )
    {
      cg::handleError( hipSetDevice( gpu )                                ) ;
      cg::handleError( hipMemcpy( dst, src, amt, hipMemcpyHostToDevice ) ) ;
    }
    
    void Cuda::copyToHost( const Cuda::Memory& src, Cuda::Data dst, Cuda::Device& gpu, unsigned amt )
    {
      cg::handleError( hipSetDevice( gpu )                                ) ;
      cg::handleError( hipMemcpy( dst, src, amt, hipMemcpyDeviceToHost ) ) ;
    }
    
    void Cuda::free( Cuda::Memory& mem, Cuda::Device& gpu )
    {
      cg::handleError( hipSetDevice( gpu ) ) ;
      cg::handleError( hipFree( mem )      ) ;
    }
    
    Cuda::Memory Cuda::createMemory( unsigned size, const Cuda::Device& gpu, Cuda::MemoryFlags flags )
    {
      return Cuda::createMemory( size, gpu ) ;
    }
    
    Cuda::Memory Cuda::createMemory( unsigned size, const Cuda::Device& gpu )
    {
      unsigned char* data ;

      cg::handleError( hipSetDevice( gpu )         ) ;
      cg::handleError( hipMalloc   ( &data, size ) ) ;
      
      return data ;
    }
  }
}
